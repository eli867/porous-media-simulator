#include "hip/hip_runtime.h"
/*

TSSD Submodel Main File:


This file will run most of the TSSD subroutines for
estimating the diffusion coefficient in the active material
by matching data from the neutron experiment.


Andre Adam.

Last Updated:

05/13/2025
*/

#include <TSSD.cuh>

int main(int argc, char **argv)
{
    // Declare structs
    options opts;
    TSSDopts oTSSD;
    meshInfo mesh;
    Migration mig;

    // TSSD Input Name

    char inputFilename[50];

    sprintf(inputFilename, "inputTSSD.txt");

    // Check if file exists

    bool fileExist = false;

    // Check if file exists

    if (FILE *TEST = fopen(inputFilename, "r"))
    {
        fclose(TEST);
        fileExist = true;
    }

    if (!fileExist)
    {
        printf("Input file not found, returning...\n");
        return 1;
    }

    // read input

    readInputGeneral(inputFilename, &opts);

    // read input TSSD
    readInputTSSD(inputFilename, &oTSSD);

    // read mig if necessary
    if (oTSSD.useMig)
    {
        bool error = readInputMig(&mig);
        if(error)
        {
            printf("Some error occured while trying to read the input file for Migration model.\n");
            printf("Exiting Now!\n");
            return 1;
        }
    }

    // print options

    if (opts.verbose)
        printTSSD(&opts, &oTSSD, &mig);


    // Load image to simulate

    char *simData;

    readImg2D(&opts, &mesh, simData);

    // set mesh parameters

    mesh.dx = oTSSD.pixelRes / opts.MeshIncreaseX;
    mesh.dy = oTSSD.pixelRes / opts.MeshIncreaseY;

    // Automatically find dt

    double maxDC = 0;

    for (int i = 0; i < opts.numDC; i++)
    {
        if (i == 0 && opts.DC[i] != 0)
            maxDC = opts.DC[i];
        else if (opts.DC[i] != 0 && opts.DC[i] > maxDC)
            maxDC = opts.DC[i];
    }

    mesh.dt = 20 * mesh.dx * mesh.dx / maxDC;
    if (oTSSD.useMig)
    {
        if (fabs(mesh.dx * maxDC * opts.charge * FARADAY / (GAS_C * mig.T) * mig.dE_dL[1]) > maxDC)
        {
            double temp = fabs(mesh.dx * maxDC * opts.charge * FARADAY / (GAS_C * mig.T) * mig.dE_dL[1]);
            mesh.dt = 20 * mesh.dx * mesh.dx / temp;
        }
    }

    if (opts.verbose)
    {
        printf("Pixel Res = %1.3e\n", mesh.dx);
        printf("Mesh DT = %1.3e\n", mesh.dt);
    }

    // if multiple methods are selected, just return an error

    if(oTSSD.useGITT + oTSSD.useLinear + oTSSD.useAnom > 1)
    {
        printf("Multiple models selected, returning...\n");
        return 1;
    }

    // Create arrays for BC's and DC's

    double *DC = (double *)malloc(sizeof(double) * mesh.nElements);
    int *BC = (int *)malloc(sizeof(int) * (mesh.numCellsY + 2) * (mesh.numCellsX + 2));
    double *BC_Value = (double *)malloc(sizeof(double) * (mesh.numCellsY + 2) * (mesh.numCellsX + 2));

    double *GITT_D;
    double *GITT_SOC;
    int nData = 100;    // just a hardcoded default, assuming I don't have more than 100 GITT points

    if (oTSSD.useGITT)
    {
        // create space for arrays
        GITT_D = (double *)malloc(sizeof(double) * nData);
        GITT_SOC = (double *)malloc(sizeof(double) * nData);
        //  set memory
        memset(GITT_D, 0, sizeof(double) * nData);
        memset(GITT_SOC, 0, sizeof(double) * nData);
        // read GITT data
        GITT_Interval(&oTSSD, GITT_SOC, GITT_D, &nData);
    }

    // initialize arrays

    memset(DC, 0, sizeof(double) * mesh.nElements);
    memset(BC, 0, sizeof(int) * (mesh.numCellsY + 2) * (mesh.numCellsX + 2));
    memset(BC_Value, 0, sizeof(double) * (mesh.numCellsY + 2) * (mesh.numCellsX + 2));

    // start an array for SOC

    double SOC = 0;
    int GITT_idx = 0;
    double POI_DC = 0;

    if (oTSSD.useGITT == 0)
        SetDC2D(&opts, &mesh, DC, simData);
    else
    {
        POI_DC = GITT_D[GITT_idx];
        SetDC_GITT(&opts, &oTSSD, &mesh, DC, simData, POI_DC);
    }

    // BC Conditions for TSSD Model

    activeSA_2D(&opts, &mesh, DC);

    SetBC_TSSD2D(&opts, &oTSSD, &mesh, BC, BC_Value);

    // Flood-Fill Bottom Start
    FloodFill2D_Bot(&mesh, BC, DC);

    // Load data to match

    /*
        Not there yet
    */

    // Allocate arrays for holding discretized equations

    double *CoeffMatrix = (double *)malloc(mesh.nElements * 5 * sizeof(double));
    double *RHS = (double *)malloc(mesh.nElements * sizeof(double));
    double *Concentration = (double *)malloc(mesh.nElements * sizeof(double));

    double *C0 = (double *)malloc(sizeof(double) * mesh.nElements);

    // initialize the memory

    memset(CoeffMatrix, 0.0, mesh.nElements * sizeof(double) * 5);
    memset(RHS, 0.0, mesh.nElements * sizeof(double));
    memset(Concentration, 0.0, mesh.nElements * sizeof(double));
    memset(C0, 0.0, sizeof(double) * mesh.nElements);

    for(int i = 0; i < mesh.nElements; i++)
    {
        if (DC[i] == 0)
            continue;
        Concentration[i] = oTSSD.C0;    // mol/m^3
        C0[i] = oTSSD.C0;               // mol/m^3
    }

    if(oTSSD.useAnom)
    {
        // populate DC array
        setDC_AnomDiff(&oTSSD, &mesh, DC, Concentration, simData);
    }

    
    // if using linear model, update

    if(oTSSD.useLinear)
    {
        SetDC_Linear(&opts, &oTSSD, &mesh, DC, simData, Concentration);
    }

    // Declare needed arrays

    double *d_Coeff = NULL;
    double *d_RHS = NULL;
    double *d_Conc = NULL;
    double *d_ConcTemp = NULL;

    // Now we confirm that there is a match in GPUs available and user expectations

    if (opts.useGPU) 
    {
        int nDevices;
        hipGetDeviceCount(&nDevices);

        if (nDevices < 1)
        {
            printf("No CUDA-capable GPU Detected! Exiting...\n");
            return 1;
        }
        else if (nDevices < opts.nGPU)
        {
            printf("User requested %d GPUs, but only %d were detected.\n", opts.nGPU, nDevices);
            printf("Proceeding with %d GPUs\n", nDevices);
            opts.nGPU = nDevices;
        }

        // Initialize the GPU arrays

        initGPU_2DSOR(&d_Coeff, &d_RHS, &d_Conc, &d_ConcTemp, &mesh);
    }

    // New discretization needed
    DiscTrans2D(&opts, &mesh, BC, BC_Value, DC, CoeffMatrix, RHS, C0);

    // Migration contribution to discretization
    if(oTSSD.useMig)
        Disc_Mig2D(CoeffMatrix, DC, RHS, C0, &opts, &mesh, &mig);

    mesh.currentTime = 0;

    int step = 0;

    double timeToCheck = oTSSD.stepSize;

    // save C(y,t)

    saveCyt(&mesh, Concentration, step);

    while (mesh.currentTime <= oTSSD.totalTime)
    {
        // if using GITT data, check for updates to DC
        if(oTSSD.useGITT)
        {
            SOC = mesh.currentTime / oTSSD.totalTime * 100;
            if (SOC >= GITT_SOC[GITT_idx + 1] && GITT_SOC[GITT_idx + 1] != 0 && GITT_D[GITT_idx + 1] != 0)
            {
                // update DC
                GITT_idx++;
                POI_DC = GITT_D[GITT_idx];
                SetDC_GITT(&opts, &oTSSD, &mesh, DC, simData, POI_DC);
                // discretize system again
                DiscTrans2D(&opts, &mesh, BC, BC_Value, DC, CoeffMatrix, RHS, C0);
                printf("Updated DC: %1.3e, Time = %1.3e, SOC = %1.3e\n", POI_DC, mesh.currentTime, SOC);
            }
            else if(mesh.currentTime != 0)
            {
                // no DC update, only update RHS
                RHS_Update2D(&mesh, BC, BC_Value, CoeffMatrix, RHS, C0);
            }
        } 
        else if(oTSSD.useLinear)
        {
            // update diffusion coefficients
            SetDC_Linear(&opts, &oTSSD, &mesh, DC, simData, Concentration);
            // discretize system again
            DiscTrans2D(&opts, &mesh, BC, BC_Value, DC, CoeffMatrix, RHS, C0);
        }
        else if(oTSSD.useAnom)
        {
            // update DC and discretize again
            setDC_AnomDiff(&oTSSD, &mesh, DC, Concentration, simData);

            DiscTrans2D(&opts, &mesh, BC, BC_Value, DC, CoeffMatrix, RHS, C0);
        }
        else
        {
            // not using GITT data
            if (mesh.currentTime != 0)
            {
                // coefficient matrix is still good, just update the RHS
                // RHS_Update2D(&mesh, BC, BC_Value, CoeffMatrix, RHS, C0);
                DiscTrans2D(&opts, &mesh, BC, BC_Value, DC, CoeffMatrix, RHS, C0);
                Disc_Mig2D(CoeffMatrix, DC, RHS, C0, &opts, &mesh, &mig);
            }
        }


        if (opts.useGPU == 0)
        {
            // CPU Solve
            omp_set_num_threads(opts.nThreads);

            GS2D_OMP(CoeffMatrix, RHS, Concentration, &opts, &mesh);
        }
        else
        {
            // GPU Solve
            if (mesh.currentTime == 0)
            {
                JI2D_SOR(CoeffMatrix, RHS, Concentration, d_Coeff,
                         d_RHS, d_Conc, d_ConcTemp, &opts, &mesh);
            }
            else
            {
                JI2D_TransientUpdate(RHS, Concentration, d_Coeff,
                                     d_RHS, d_Conc, d_ConcTemp, &opts, &mesh);
            }
        }

        // Update time
        mesh.currentTime += mesh.dt;

        // Copy new concentration into C0
        memcpy(C0, Concentration, sizeof(double) * mesh.nElements);

        // save data if necessary
        if (mesh.currentTime > timeToCheck)
        {
            timeToCheck += oTSSD.stepSize;
            step++;

            saveCyt(&mesh, Concentration, step);
            if (opts.verbose)
                printf("Time = %1.3e\n", mesh.currentTime);
            
            // check NaN's
            for(int i = 0; i < mesh.nElements; i++)
            {
                if(Concentration[i] != Concentration[i])
                {
                    printf("Found NaN at %d, time %1.3e\n", i, mesh.currentTime);
                    return 1;
                }
            }
        }
    }

    printCandF(&opts, &oTSSD, &mesh, DC, Concentration);

    // Pick simulations that match the concentration profile by some metric

    // Interpolate the actual diffusion coefficient

    // Simulate the new concentration distribution based on theory of anomalous diffusion
    // and using the previous value statically.

    // Assess these two results, interpolate, find a new average coefficient.
    /*
        If new coefficient is similar to anomalous diffusion theory, then don't use it.

        If new coefficient is better, then use it to simulate concentration map again.

        NOTES:
            - what metric to use ?
            - What is good enough ?
    */

    // repeat these steps for a full charge cycle, full discharge cycle.

    /*
        Data to be saved:
            - save average Lithium concentration in y-direction
            - save Li concentration maps every 5 minutes.
            - Let's use a small domain for this simulation.
    */

    // Manage GPU Memory (if applicable)

    if (opts.useGPU)
    {
        unInitGPU_SOR(&d_Coeff, &d_RHS, &d_Conc, &d_ConcTemp);
    }

    // Memory management

    free(CoeffMatrix);
    free(Concentration);
    free(C0);
    free(RHS);

    free(BC);
    free(BC_Value);
    free(DC);

    return 0;
}